#include "hip/hip_runtime.h"
#include <backprop_tools/operations/hip/hip_runtime.h>


#include <backprop_tools/rl/environments/multirotor/operations_cpu.h>
#include "parameters.h"
namespace bpt = BACKPROP_TOOLS_NAMESPACE_WRAPPER ::backprop_tools;

#include <iostream>
#include <chrono>
#include <cassert>

using T = float;
constexpr T DT = 0.01;
constexpr size_t N_BLOCKS = 64;
constexpr size_t N_BLOCKS_CPU = 1;
constexpr size_t N_THREADS = 128;
constexpr size_t N_THREADS_CPU = 1;
constexpr size_t N_ITERATIONS = 1000000;

using DEVICE_GPU = bpt::devices::CUDA<bpt::devices::DefaultCUDASpecification>;
using DEVICE_CPU = bpt::devices::CPU<bpt::devices::DefaultCPUSpecification>;

using TI_GPU = DEVICE_CPU::index_t;
using TI_CPU = DEVICE_CPU::index_t;

using penv = parameters_fast_learning::environment<T, TI_GPU>;
using ENVIRONMENT = typename penv::ENVIRONMENT;
using STATE = typename ENVIRONMENT::State;


template <TI_GPU T_N_BLOCKS, TI_GPU T_BLOCK_DIM, TI_GPU T_N_ITERATIONS>
struct SimulateParallelSpec{
    static constexpr TI_GPU N_BLOCKS = T_N_BLOCKS;
    static constexpr TI_GPU BLOCK_DIM = T_BLOCK_DIM;
    static constexpr TI_GPU N_ITERATIONS = T_N_ITERATIONS;
};

template <typename DEVICE, typename SPEC, typename SPEC_SIMULATE>
void simulate_sequential(DEVICE& device, const bpt::rl::environments::Multirotor<SPEC>* envs, const typename bpt::rl::environments::Multirotor<SPEC>::State* states_input, typename bpt::rl::environments::Multirotor<SPEC>::State* next_states_output, const SPEC_SIMULATE) {
    using ENVIRONMENT = bpt::rl::environments::Multirotor<SPEC>;
    using STATE = typename ENVIRONMENT::State;
    using TI = typename DEVICE::index_t;
    for(TI block_i = 0; block_i < SPEC_SIMULATE::N_BLOCKS; block_i++){
        for(TI thread_i = 0; thread_i < SPEC_SIMULATE::BLOCK_DIM; thread_i++){
            const TI full_id = block_i * SPEC_SIMULATE::BLOCK_DIM + thread_i;
            const auto& env = envs[full_id];
            STATE state;
            STATE next_state;
            state = states_input[full_id];
            for(TI iteration_i=0; iteration_i<SPEC_SIMULATE::N_ITERATIONS; iteration_i++){
                T action[ENVIRONMENT::ACTION_DIM];
//        evaluate(policy, state, action);
                for(TI action_i=0; action_i<ENVIRONMENT::ACTION_DIM; action_i++){
                    action[action_i] = 0;
                }
                bpt::utils::integrators::rk4<DEVICE, T, typename SPEC::PARAMETERS, STATE, ENVIRONMENT::ACTION_DIM, bpt::rl::environments::multirotor::multirotor_dynamics_dispatch<DEVICE, typename SPEC::T, typename SPEC::PARAMETERS, STATE>>(device, env.parameters, state, action, env.parameters.integration.dt, next_state);
                state = next_state;
            }
            next_states_output[full_id] = state;
        }
    }
}

template <typename DEVICE, typename SPEC, typename SPEC_SIMULATE>
__global__ void
__launch_bounds__(SPEC_SIMULATE::BLOCK_DIM)//, minBlocksPerMultiprocessor, maxBlocksPerCluster)
simulate_parallel(DEVICE& device, const bpt::rl::environments::Multirotor<SPEC>* envs, const typename bpt::rl::environments::Multirotor<SPEC>::State* states_input, typename bpt::rl::environments::Multirotor<SPEC>::State* next_states_output, const SPEC_SIMULATE) {
    using ENVIRONMENT = bpt::rl::environments::Multirotor<SPEC>;
    using STATE = typename ENVIRONMENT::State;
    using TI = typename DEVICE::index_t;
    const TI full_id = blockIdx.x * blockDim.x + threadIdx.x;
    const TI thread_id = threadIdx.x;
    const TI block_id = blockIdx.x;
    __shared__ ENVIRONMENT env;
    if(thread_id == 0){
        env = envs[block_id];
    }
    __syncthreads();
    STATE state;
    STATE next_state;
    state = states_input[full_id];
    for(TI iteration_i=0; iteration_i<SPEC_SIMULATE::N_ITERATIONS; iteration_i++){
        T action[ENVIRONMENT::ACTION_DIM];
//        evaluate(policy, state, action);
        for(TI action_i=0; action_i<ENVIRONMENT::ACTION_DIM; action_i++){
            action[action_i] = 0;
        }
        bpt::utils::integrators::rk4<DEVICE, T, typename SPEC::PARAMETERS, STATE, ENVIRONMENT::ACTION_DIM, bpt::rl::environments::multirotor::multirotor_dynamics_dispatch<DEVICE, typename SPEC::T, typename SPEC::PARAMETERS, STATE>>(device, env.parameters, state, action, env.parameters.integration.dt, next_state);
        state = next_state;
    }
    next_states_output[full_id] = state;
}

int main(void) {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "\tName: " << prop.name << std::endl;
        std::cout << "\tCompute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "\tNumber of SMs: " << prop.multiProcessorCount << std::endl;
        std::cout << "\tRegisters per Multiprocessor: " << prop.regsPerMultiprocessor << std::endl;
        std::cout << "\tMax threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
    }
    TI_CPU chosen_device = 0;
    hipSetDevice(chosen_device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, chosen_device);


    DEVICE_CPU device_cpu;
    DEVICE_GPU device_gpu;

    STATE initial_states[N_BLOCKS][N_THREADS];
    STATE final_states_gpu[N_BLOCKS][N_THREADS];
    STATE final_states_cpu[N_BLOCKS][N_THREADS];
    ENVIRONMENT envs[N_BLOCKS][N_THREADS];
    for(TI_CPU block_i=0; block_i<N_BLOCKS; block_i++){
        for(TI_CPU thread_i=0; thread_i<N_THREADS; thread_i++){
            envs[block_i][thread_i].parameters = penv::parameters;
            envs[block_i][thread_i].parameters.integration.dt = DT;
            bpt::initial_state(device_cpu, envs[block_i][thread_i], initial_states[block_i][thread_i]);
        }
    }

    {
        auto start = std::chrono::high_resolution_clock::now();
        simulate_sequential(device_cpu, &envs[0][0], &initial_states[0][0], &final_states_cpu[0][0], SimulateParallelSpec<N_BLOCKS_CPU, N_THREADS_CPU, N_ITERATIONS>{});
        auto end = std::chrono::high_resolution_clock::now();

        double elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

        std::cout << "Simulation time (CPU):  " << elapsedTime << " ms (" << N_BLOCKS_CPU * N_THREADS_CPU * N_ITERATIONS / (elapsedTime / 1000.0) / 1e6 << " Msteps/s)" << std::endl;
    }
    {
        ENVIRONMENT *d_envs;
        STATE* d_states;
        STATE* d_next_states;

        hipMalloc((void **)&d_envs, N_BLOCKS * sizeof(ENVIRONMENT));
        hipMemcpy(d_envs, envs, N_BLOCKS * sizeof(ENVIRONMENT), hipMemcpyHostToDevice);

        hipMalloc((void **)&d_states, N_BLOCKS * N_THREADS * sizeof(STATE));
        hipMemcpy(d_states, &initial_states, N_BLOCKS * N_THREADS * sizeof(STATE), hipMemcpyHostToDevice);

        hipMemcpy(&initial_states, d_states, N_BLOCKS * N_THREADS * sizeof(STATE), hipMemcpyDeviceToHost);

        hipMalloc((void **)&d_next_states, N_BLOCKS * N_THREADS * sizeof(STATE));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        dim3 grid(N_BLOCKS);
        dim3 threadsPerBlock(N_THREADS);
        simulate_parallel<<<grid, threadsPerBlock>>>(device_gpu, d_envs, d_states, d_next_states, SimulateParallelSpec<N_BLOCKS, N_THREADS, N_ITERATIONS>{});
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        auto err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
        }
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipDeviceSynchronize();
        hipMemcpy(final_states_gpu, d_next_states, N_BLOCKS * N_THREADS * sizeof(STATE), hipMemcpyDeviceToHost);
        hipFree(d_envs); hipFree(d_states); hipFree(d_next_states);
        hipDeviceSynchronize();

        std::cout << "Simulation time (GPU):  " << elapsedTime << " ms (" << N_BLOCKS * N_THREADS * N_ITERATIONS / (elapsedTime / 1000.0) / 1e6 << " Msteps/s, " << N_BLOCKS * N_THREADS * N_ITERATIONS * DT / (elapsedTime / 1000.0) / (365 * 24 * 3600) << " Years/s)" << std::endl;
    }

//    // Copy a & b from the host to the device
//    T diff = 0;
//    for(int i=N_BLOCKS-N_BLOCKS_EVAL; i < N_BLOCKS; i++){
//        for(int j=N_THREADS-N_THREADS_EVAL; j < N_THREADS; j++) {
//            for (int k = 0; k < STATE_DIM; k++) {
//                diff += std::fabs(state_cpu[i][j][k] - state_gpu[i][j][k]);
//            }
//        }
//    }
//    std::cout << "Average diff (cpu-gpu): " << diff/(T)(N_BLOCKS * N_THREADS) << std::endl;
//
//    std::cout << "Final state:" << std::endl;
//    std::cout.precision(17);
//    for(int i=0; i < STATE_DIM; i++){
//        std::cout << state_cpu[N_BLOCKS-1][N_THREADS-1][i] << " ";
//    }
//    std::cout << std::endl;
//
//    std::cout << "Final state comparison cpu <-> gpu:" << std::endl;
//    for(int i=0; i < STATE_DIM; i++){
//        std::cout << state_cpu[N_BLOCKS-1][N_THREADS-1][i] - state_gpu[N_BLOCKS-1][N_THREADS-1][i] << " ";
//    }
//    std::cout << std::endl;
//
//    std::cout << "Final state comparison gpu <-> jax:" << std::endl;
//    T acc = 0;
//    for(int i=0; i < N_BLOCKS; i++){
//        for(int j=0; j < N_THREADS; j++) {
//            for (int i = 0; i < STATE_DIM; i++) {
//                T diff = state_gpu[N_BLOCKS - 1][N_THREADS - 1][i] - expected_state2[i];
//                acc += std::abs(diff);
//                if(i == 0 && j == 0){
//                    std::cout << diff << " ";
//                }
//            }
//        }
//    }
//    std::cout << std::endl;
//    std::cout << "Final state comparison gpu <-> jax (cumulative): " << acc / ((T)N_BLOCKS * N_THREADS) << std::endl;


    return 0;
}
